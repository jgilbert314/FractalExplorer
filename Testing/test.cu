

#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <iostream>  // Input/Output
#include <sstream>

using namespace std; // Allow implicit use of standard library

#include <chrono>
using namespace chrono;

//         123456789
#define N 500000000

__global__
void vector_add(float *out, float *a, float *b, int n) {
    for (int itr = 0; itr < n; itr++) {
        out[itr] = a[itr] + b[itr];
    }
}

void vector_addC(float *out, float *a, float *b, int n) {
    for (int itr = 0; itr < n; itr++) {
        out[itr] = a[itr] + b[itr];
    }
}


int main(int argc, char *argv[]) {


    auto t0 = high_resolution_clock::now();
    auto t1 = high_resolution_clock::now();

    float *a, *b, *out;

    // Memory
    uint N_f = sizeof(float);
    a = (float*)malloc(N_f*N);
    b = (float*)malloc(N_f*N);
    out = (float*)malloc(N_f*N);
    uint mem_tot = 3*N_f*N / 1e6;

    // Initialize
    for (int itr = 0; itr < N; itr++) {
        a[itr] = 1.0f;
        b[itr] = 2.0f;
    }

    t0 = high_resolution_clock::now();
    vector_add<<<1, 1>>>(out, a, b, N);
    t1 = high_resolution_clock::now();
    duration<double> countG = (t1 - t0);
    
    t0 = high_resolution_clock::now();
    vector_addC(out, a, b, N);
    t1 = high_resolution_clock::now();
    duration<double> countC = (t1 - t0);

    delete [] a;
    delete [] b;
    delete [] out;

    cout << "Mem: " << mem_tot << endl;
    cout << "CPU: " << countC.count() << endl;
    cout << "GPU: " << countG.count() << endl;
    


    return 0;
}